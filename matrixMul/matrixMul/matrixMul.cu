#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <memory>
#include <hip/hip_runtime.h>

const short m = 3;
const short n = 4;
const short k = 2;

const dim3 blockNum(1, 1, 1);
const dim3 threadNum(8, 8, 1);

template<typename T>
void initalValue(T& a, const short length, const float value)
{
	for (short i = 0; i < length; i++)
	{
		a[i] = value;
	}
}

template<typename T>
void checkValue(T a, const short length, const short col)
{
	for (short i = 0; i < length; i++)
	{
		printf("%.3f ", a[i]);
		if ((i - 1) % col == 0)
		{
			printf("\n");
		}
	}
	printf("\n");
}

__global__ void matrixMul(const float* a, const float* b, float* c, const short m, const short n, const short k)
{
	const short col = threadIdx.x + blockIdx.x * blockDim.x;
	const short row = threadIdx.y + blockIdx.y * blockDim.y;

	float c_temp = 0.0;
	if (row < m  && col < k)
	{
		for (short i = 0; i < n; i++)
		{
			c_temp += a[row * n + i] * b[i * k + col];
		}
		c[row * k + col] = c_temp;
	}	
}

__global__ void matrixMulWithSharedMemory(const float* a, const float* b, float* c, const short m, const short n, const short k)
{
	const short col = threadIdx.x + blockIdx.x * blockDim.x;
	const short row = threadIdx.y + blockIdx.y * blockDim.y;

	__shared__ float VALUE[6];

	if (row < m && col < k)
	{
		for (short i = 0; i < n; i++)
		{
			VALUE[row * k + col] += a[row * n + i] * b[i * k + col];
		}
		__syncthreads();
	}
	c[row * k + col] = VALUE[row * k + col];
}

int main()
{
	// initial host memory 
	float* h_a = nullptr;		// m * n
	float* h_b = nullptr;		// n * k
	float* h_c = nullptr;		// m * k

	h_a = (float*)malloc(m * n * sizeof(float));
	h_b = (float*)malloc(n * k * sizeof(float));
	h_c = (float*)malloc(m * k * sizeof(float));

	if (h_a == nullptr || h_b == nullptr || h_c == nullptr)
	{
		printf("memroy malloc error with code 001!\n");
		exit(EXIT_FAILURE);
	}

	// initial host matrix value
	initalValue(h_a, m * n, 1.0);
	initalValue(h_b, n * k, 2.0);	

	// value check
	checkValue(h_a, m * n, n);
	checkValue(h_b, n * k, k);

	// initial device memory
	float* d_a = nullptr;		// m * n 
	float* d_b = nullptr;		// n * k
	float* d_c = nullptr;		// m * k

	hipMalloc((void**)&d_a, m * n * sizeof(float));
	hipMalloc((void**)&d_b, n * k * sizeof(float));
	hipMalloc((void**)&d_c, m * k * sizeof(float));

	// copy data from host to device
	hipMemcpy(d_a, h_a, m * n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, n * k * sizeof(float), hipMemcpyHostToDevice);

	// calculate matrixs
	matrixMul << <blockNum, threadNum >> > (d_a, d_b, d_c, m, n, k);

	hipMemcpy(h_c, d_c, m * k * sizeof(float), hipMemcpyDeviceToHost);
	checkValue(h_c, m * k, k);

	// calculate matrix with shared memory
	hipMemset(d_c, 0, m * k * sizeof(float));
	memset(h_c, 0, m * k * sizeof(float));

	matrixMulWithSharedMemory << <blockNum, threadNum >> > (d_a, d_b, d_c, m, n, k);

	// copy calculate result from device to host
	hipMemcpy(h_c, d_c, m * k * sizeof(float), hipMemcpyDeviceToHost);
	checkValue(h_c, m * k, k);

	free(h_a);
	free(h_b);
	free(h_c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);


	system("pause");
	return 0;
}